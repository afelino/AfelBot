﻿// Jetson Nano.cpp: определяет точку входа для приложения.

#include <evhttp.h>
#include <memory>

#include "main.h"
#include "AfelBotConfig.h"

using namespace std;

int main()
{
	// Выводим текущую версию программы.
	cout << "AfelBot version " << AFELBOT_VERSION << endl;

    //Специальная инициализация для винды.
    #ifdef _WIN32
        WSADATA wsa_data;
        WSAStartup(0x0201, &wsa_data);
    #endif
        
    if (!event_init()) {
        cerr << "Ошибка инициализации LibEvent." << endl;
        return -1;
    }
    
    char const SrvAddress[] = "127.0.0.1";
    uint16_t SrvPort = 5555;
    unique_ptr<evhttp, decltype(&evhttp_free)> Server(evhttp_start(SrvAddress, SrvPort), &evhttp_free);
    if (!Server){
        cerr << "Не удалось запустить сервер HTTP" << endl;
        return -1;
    }
    void (*OnReq)(evhttp_request * req, void*) = [](evhttp_request* req, void*)
    {
        auto* OutBuf = evhttp_request_get_output_buffer(req);
        if (!OutBuf) {
            return;
        }
            
        evbuffer_add_printf(OutBuf, "<html><body><center><h1>Привет, мир!</h1></center></body></html>");
        evhttp_send_reply(req, HTTP_OK, "", OutBuf);
    };
    
    evhttp_set_gencb(Server.get(), OnReq, nullptr);
    if (event_dispatch() == -1)
    {
        cerr << "Не удалось запустить цикл обработки событий." << endl;
        return -1;
    }

	return 0;
}
